#include "hip/hip_runtime.h"
/*
 * Author: Brian Bowden
 * Date: 5/12/14
 *
 * This is the parallel version of SHA-3.
 */
 
#include <stdlib.h>
#include <stdio.h>
#include <stdint.h>

void gpu_init();
void runBenchmarks();
char *read_in_messages();
int gcd(int a, int b);

// updated message the gpu_init() function
int clock_speed;
int number_multi_processors;
int number_blocks;
int number_threads;
int max_threads_per_mp;

int num_messages;
const int digest_size = 256;
const int digest_size_bytes = digest_size / 8;
const size_t str_length = 7;	//change for different sizes

hipEvent_t start, stop;

#define ROTL64(x, y) (((x) << (y)) | ((x) >> (64 - (y))))

__device__ const char *chars = 
    " !\"#$%&\'()*+'-./0123456789:;<=>?@ABCDEFGHIJKLMOPQRSTUVWXYZ[\\]^_`abcdefghijklmnopqrstuvwxyz{|}~";
	
__device__ const uint64_t RC[24] = {
    0x0000000000000001, 0x0000000000008082, 0x800000000000808a,
    0x8000000080008000, 0x000000000000808b, 0x0000000080000001,
    0x8000000080008081, 0x8000000000008009, 0x000000000000008a,
    0x0000000000000088, 0x0000000080008009, 0x000000008000000a,
    0x000000008000808b, 0x800000000000008b, 0x8000000000008089,
    0x8000000000008003, 0x8000000000008002, 0x8000000000000080, 
    0x000000000000800a, 0x800000008000000a, 0x8000000080008081,
    0x8000000000008080, 0x0000000080000001, 0x8000000080008008
};

__device__ const int r[24] = {
    1,  3,  6,  10, 15, 21, 28, 36, 45, 55, 2,  14, 
    27, 41, 56, 8,  25, 43, 62, 18, 39, 61, 20, 44
};

__device__ const int piln[24] = {
    10, 7,  11, 17, 18, 3, 5,  16, 8,  21, 24, 4, 
    15, 23, 19, 13, 12, 2, 20, 14, 22, 9,  6,  1 
};

__device__ void generate_message(char *message, uint64_t tid, int *str_len)
{
	int len = 0;
	const int num_chars = 94;
	char str[21];
	while (tid > 0)
	{
		str[len++] = chars[tid % num_chars];
		tid /= num_chars;
	}
	
	str[len] = '\0';
	memcpy(message, str, len + 1);
	*str_len = len;
}

__device__ int compare_hash(unsigned char *target, unsigned char *hash, int length)
{
	for (int i = 0; i < length; i++)
	{
		if (target[i] != hash[i])
		    return 0;
	}
	
	return 1;
}

__device__ void keccak256(uint64_t state[25])
{
    uint64_t temp, C[5];
	int j;
	
    for (int i = 0; i < 24; i++) {
        // Theta
		// for i = 0 to 5 
		//    C[i] = state[i] ^ state[i + 5] ^ state[i + 10] ^ state[i + 15] ^ state[i + 20];
		C[0] = state[0] ^ state[5] ^ state[10] ^ state[15] ^ state[20];
		C[1] = state[1] ^ state[6] ^ state[11] ^ state[16] ^ state[21];
		C[2] = state[2] ^ state[7] ^ state[12] ^ state[17] ^ state[22];
		C[3] = state[3] ^ state[8] ^ state[13] ^ state[18] ^ state[23];
		C[4] = state[4] ^ state[9] ^ state[14] ^ state[19] ^ state[24];
		
		// for i = 0 to 5
		//     temp = C[(i + 4) % 5] ^ ROTL64(C[(i + 1) % 5], 1);
		//     for j = 0 to 25, j += 5
		//          state[j + i] ^= temp;
		temp = C[4] ^ ROTL64(C[1], 1);
		state[0] ^= temp;
		state[5] ^= temp;
		state[10] ^= temp;
		state[15] ^= temp;
		state[20] ^= temp;
		
		temp = C[0] ^ ROTL64(C[2], 1);
		state[1] ^= temp;
		state[6] ^= temp;
		state[11] ^= temp;
		state[16] ^= temp;
		state[21] ^= temp;
		
		temp = C[1] ^ ROTL64(C[3], 1);
		state[2] ^= temp;
		state[7] ^= temp;
		state[12] ^= temp;
		state[17] ^= temp;
		state[22] ^= temp;
		
		temp = C[2] ^ ROTL64(C[4], 1);
		state[3] ^= temp;
		state[8] ^= temp;
		state[13] ^= temp;
		state[18] ^= temp;
		state[23] ^= temp;
		
		temp = C[3] ^ ROTL64(C[0], 1);
		state[4] ^= temp;
		state[9] ^= temp;
		state[14] ^= temp;
		state[19] ^= temp;
		state[24] ^= temp;
		
        // Rho Pi
		// for i = 0 to 24
		//     j = piln[i];
		//     C[0] = state[j];
		//     state[j] = ROTL64(temp, r[i]);
		//     temp = C[0];
		temp = state[1];
		j = piln[0];
		C[0] = state[j];
		state[j] = ROTL64(temp, r[0]);
		temp = C[0];
		
		j = piln[1];
		C[0] = state[j];
		state[j] = ROTL64(temp, r[1]);
		temp = C[0];
		
		j = piln[2];
		C[0] = state[j];
		state[j] = ROTL64(temp, r[2]);
		temp = C[0];
		
		j = piln[3];
		C[0] = state[j];
		state[j] = ROTL64(temp, r[3]);
		temp = C[0];
		
		j = piln[4];
		C[0] = state[j];
		state[j] = ROTL64(temp, r[4]);
		temp = C[0];
		
		j = piln[5];
		C[0] = state[j];
		state[j] = ROTL64(temp, r[5]);
		temp = C[0];
		
		j = piln[6];
		C[0] = state[j];
		state[j] = ROTL64(temp, r[6]);
		temp = C[0];
		
		j = piln[7];
		C[0] = state[j];
		state[j] = ROTL64(temp, r[7]);
		temp = C[0];
		
		j = piln[8];
		C[0] = state[j];
		state[j] = ROTL64(temp, r[8]);
		temp = C[0];
		
		j = piln[9];
		C[0] = state[j];
		state[j] = ROTL64(temp, r[9]);
		temp = C[0];
		
		j = piln[10];
		C[0] = state[j];
		state[j] = ROTL64(temp, r[10]);
		temp = C[0];
		
		j = piln[11];
		C[0] = state[j];
		state[j] = ROTL64(temp, r[11]);
		temp = C[0];
		
		j = piln[12];
		C[0] = state[j];
		state[j] = ROTL64(temp, r[12]);
		temp = C[0];
		
		j = piln[13];
		C[0] = state[j];
		state[j] = ROTL64(temp, r[13]);
		temp = C[0];
		
		j = piln[14];
		C[0] = state[j];
		state[j] = ROTL64(temp, r[14]);
		temp = C[0];
		
		j = piln[15];
		C[0] = state[j];
		state[j] = ROTL64(temp, r[15]);
		temp = C[0];
		
		j = piln[16];
		C[0] = state[j];
		state[j] = ROTL64(temp, r[16]);
		temp = C[0];
		
		j = piln[17];
		C[0] = state[j];
		state[j] = ROTL64(temp, r[17]);
		temp = C[0];
		
		j = piln[18];
		C[0] = state[j];
		state[j] = ROTL64(temp, r[18]);
		temp = C[0];
		
		j = piln[19];
		C[0] = state[j];
		state[j] = ROTL64(temp, r[19]);
		temp = C[0];
		
		j = piln[20];
		C[0] = state[j];
		state[j] = ROTL64(temp, r[20]);
		temp = C[0];
		
		j = piln[21];
		C[0] = state[j];
		state[j] = ROTL64(temp, r[21]);
		temp = C[0];
		
		j = piln[22];
		C[0] = state[j];
		state[j] = ROTL64(temp, r[22]);
		temp = C[0];
		
		j = piln[23];
		C[0] = state[j];
		state[j] = ROTL64(temp, r[23]);
		temp = C[0];

        //  Chi
		// for j = 0 to 25, j += 5
		//     for i = 0 to 5
		//         C[i] = state[j + i];
		//     for i = 0 to 5
		//         state[j + 1] ^= (~C[(i + 1) % 5]) & C[(i + 2) % 5];
		C[0] = state[0];
		C[1] = state[1];
		C[2] = state[2];
		C[3] = state[3];
		C[4] = state[4];
			
		state[0] ^= (~C[1]) & C[2];
		state[1] ^= (~C[2]) & C[3];
		state[2] ^= (~C[3]) & C[4];
		state[3] ^= (~C[4]) & C[0];
		state[4] ^= (~C[0]) & C[1];
		
		C[0] = state[5];
		C[1] = state[6];
		C[2] = state[7];
		C[3] = state[8];
		C[4] = state[9];
			
		state[5] ^= (~C[1]) & C[2];
		state[6] ^= (~C[2]) & C[3];
		state[7] ^= (~C[3]) & C[4];
		state[8] ^= (~C[4]) & C[0];
		state[9] ^= (~C[0]) & C[1];
		
		C[0] = state[10];
		C[1] = state[11];
		C[2] = state[12];
		C[3] = state[13];
		C[4] = state[14];
			
		state[10] ^= (~C[1]) & C[2];
		state[11] ^= (~C[2]) & C[3];
		state[12] ^= (~C[3]) & C[4];
		state[13] ^= (~C[4]) & C[0];
		state[14] ^= (~C[0]) & C[1];

		C[0] = state[15];
		C[1] = state[16];
		C[2] = state[17];
		C[3] = state[18];
		C[4] = state[19];
			
		state[15] ^= (~C[1]) & C[2];
		state[16] ^= (~C[2]) & C[3];
		state[17] ^= (~C[3]) & C[4];
		state[18] ^= (~C[4]) & C[0];
		state[19] ^= (~C[0]) & C[1];
		
		C[0] = state[20];
		C[1] = state[21];
		C[2] = state[22];
		C[3] = state[23];
		C[4] = state[24];
			
		state[20] ^= (~C[1]) & C[2];
		state[21] ^= (~C[2]) & C[3];
		state[22] ^= (~C[3]) & C[4];
		state[23] ^= (~C[4]) & C[0];
		state[24] ^= (~C[0]) & C[1];
		
        //  Iota
        state[0] ^= RC[i];
    }
}

__device__ void keccak(const char *message, int message_len, unsigned char *output, int output_len)
{
    uint64_t state[25];    
    uint8_t temp[144];
    int rsize = 136;
    int rsize_byte = 17;
    
    memset(state, 0, sizeof(state));

    for ( ; message_len >= rsize; message_len -= rsize, message += rsize) {
        for (int i = 0; i < rsize_byte; i++) {
            state[i] ^= ((uint64_t *) message)[i];
		}
        keccak256(state);
    }
    
    // last block and padding
    memcpy(temp, message, message_len);
    temp[message_len++] = 1;
    memset(temp + message_len, 0, rsize - message_len);
    temp[rsize - 1] |= 0x80;

    for (int i = 0; i < rsize_byte; i++) {
        state[i] ^= ((uint64_t *) temp)[i];
	}

    keccak256(state);
    memcpy(output, state, output_len);
}

__global__ void benchmark(const char *messages, unsigned char *output, int num_messages)
{
	const int str_len = 6;
	const int output_len = 32;
	int tid = threadIdx.x + (blockIdx.x * blockDim.x);
	int num_threads = blockDim.x * gridDim.x;
	
	for (; tid < num_messages; tid += num_threads)
	{
		keccak(&messages[tid * str_len], str_len, &output[tid * output_len], output_len);
	}
}

// hash length is 256 bits
__global__ void brute_force_single(unsigned char *hash, char *message, int *done, uint64_t starting_tid)
{
	int str_len;
	const int output_len = 32;
	int tid = threadIdx.x + (blockIdx.x * blockDim.x);
	unsigned char output[output_len];
	char current_message[21];
	
	__shared__ unsigned char s_hash[32];
	if (threadIdx.x < output_len)
	{
		s_hash[threadIdx.x] = hash[threadIdx.x];
	}
	__syncthreads();
		
	generate_message(&current_message[0], tid + starting_tid, &str_len);
	keccak(&current_message[0], str_len, &output[0], output_len);
	if (compare_hash(&s_hash[0], &output[0], output_len))
	{
		memcpy(message, current_message, str_len + 1);
		done[0] = 1;
	}
}

/**
 * Initializes the global variables by calling the hipGetDeviceProperties().
 */
void gpu_init()
{
    hipDeviceProp_t device_prop;
    int device_count, block_size;

    hipGetDeviceCount(&device_count);
    if (device_count != 1) {
        printf("Only want to test a single GPU, exiting...\n");
        exit(EXIT_FAILURE);
    }

    if (hipGetDeviceProperties(&device_prop, 0) != hipSuccess) {
        printf("Problem getting properties for device, exiting...\n");
        exit(EXIT_FAILURE);
    } 

    number_threads = device_prop.maxThreadsPerBlock;
    number_multi_processors = device_prop.multiProcessorCount;
    max_threads_per_mp = device_prop.maxThreadsPerMultiProcessor;
    block_size = (max_threads_per_mp / gcd(max_threads_per_mp, number_threads));
    number_threads = max_threads_per_mp / block_size;
    number_blocks = block_size * number_multi_processors;
    clock_speed = (int) (device_prop.memoryClockRate * 1000 * 1000);    // convert from GHz to hertz
}

int gcd(int a, int b) {
    return (a == 0) ? b : gcd(b % a, a);
}

/*
 * Opens a file name and reads all the Strings into an array of Strings.
 */
char *read_in_messages(char *file_name)
{
	FILE *f;
	if(!(f = fopen(file_name, "r")))
    {
        printf("Error opening file %s", file_name);
        exit(1);
    }

	char *messages = (char *) malloc(sizeof(char) * num_messages * str_length);
	if (messages == NULL)
	{
	    perror("Error allocating memory for list of Strings.\n");
        exit(1);
	}
	
	int index = 0;
	char buf[10];
	while(1)
	{
		if (fgets(buf, str_length + 1, f) == NULL)
		    break;
		buf[strlen(buf) - 1] = '\0';
		memcpy(&messages[index], buf, str_length);
		index += str_length - 1;
	}
	
	return messages;
}

/*
 * Runs the benchmark for the SHA-3 GPU versions.
 */
void runBenchmarks(char *file_name)
{
	float h_to_d_time = 0.0;
	float comp_time = 0.0;
	float d_to_h_time = 0.0;
	float total_time = 0.0;
    float elapsed_time;
	int hashes_per_sec;
	
	size_t array_size = sizeof(char) * str_length * num_messages;
	size_t output_size = digest_size_bytes * num_messages;
	
	// Allocate host arrays
    char *h_messages = read_in_messages(file_name);
	unsigned char *h_output = (unsigned char *) malloc(output_size);

	char *d_messages;
	unsigned char *d_output;
	
    // Allocate device arrays
    hipMalloc((void**) &d_messages, array_size);
	hipMalloc((void**) &d_output, output_size);
	
	int number_runs = 25;
    // Copy Strings from host to device arrays
    for (int j = 0; j < number_runs; j++)
	{
		hipEventRecord(start, 0);
		hipMemcpy(d_messages, h_messages, array_size, hipMemcpyHostToDevice);
		hipEventRecord(stop, 0);
		hipEventSynchronize(start);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&elapsed_time, start, stop);
		h_to_d_time += elapsed_time;

		hipEventRecord(start, 0);
		benchmark<<<number_blocks, number_threads>>>(d_messages, d_output, num_messages);
		hipEventRecord(stop, 0);
        hipEventSynchronize(start);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsed_time, start, stop);
        comp_time += elapsed_time;
	
		// Copy hashes from device to host arrays
		hipEventRecord(start, 0);
		hipMemcpy(h_output, d_output, array_size, hipMemcpyDeviceToHost);
		hipEventRecord(stop, 0);
		hipEventSynchronize(start);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&elapsed_time, start, stop);
		d_to_h_time += elapsed_time;
	}
	
	// averages the time over the number of runs and converts it from ms to sec
	h_to_d_time /= (number_runs * 1000);
	comp_time /= (number_runs * 1000);
	d_to_h_time /= (number_runs * 1000);
	total_time = h_to_d_time + comp_time + d_to_h_time;
	hashes_per_sec = num_messages / total_time;
	
	printf("Memory Transfer Time Host -> Device\temp%0.3g sec\n", h_to_d_time);
	printf("Computation Time\temp\temp\temp%0.3g sec\n", comp_time);
	printf("Memory Transfer Time Device -> Host\temp%0.3g sec\n", d_to_h_time);
	printf("Total Time\temp\temp\temp\temp%0.3g sec\n", total_time);
	printf("%lu hashes/sec\n", hashes_per_sec);
	
	for (int i = 0; i < 10; i++)
	{
		printf("%d : ", i);
		for (int j = 0; j < digest_size_bytes; j++)
		{
			printf("%02x", h_output[j + (i * digest_size_bytes)]);
		}
		printf("\n");
	}
	
	// Free arrays from memory
    free(h_messages);
	free(h_output);
    hipFree(d_messages);
	hipFree(d_output);
}

void find_message()
{
	float h_to_d_time = 0.0;
	float comp_time = 0.0;
	float d_to_h_time = 0.0;
	float total_time = 0.0;
	size_t max_str_size = 11 * sizeof(char);
	size_t digest_str_size = digest_size_bytes * sizeof(unsigned char);
	
	char hash[65] = "e6b6ca9b98ea0c1b64bee9382438c8c99f35bc4d680bcca9f2db31a577915fe4";
	unsigned char h_hash[digest_size_bytes + 1];
	sscanf(hash, "%02hhx%02hhx%02hhx%02hhx%02hhx%02hhx%02hhx%02hhx%02hhx%02hhx%02hhx%02hhx%02hhx%02hhx%02hhx%02hhx%02hhx%02hhx%02hhx%02hhx%02hhx%02hhx%02hhx%02hhx%02hhx%02hhx%02hhx%02hhx%02hhx%02hhx%02hhx%02hhx",
		&h_hash[0], &h_hash[1], &h_hash[2], &h_hash[3], &h_hash[4], &h_hash[5], &h_hash[6], &h_hash[7], &h_hash[8], &h_hash[9], &h_hash[10], &h_hash[11], &h_hash[12], &h_hash[13], &h_hash[14], &h_hash[15], 
		&h_hash[16], &h_hash[17], &h_hash[18], &h_hash[19], &h_hash[20], &h_hash[21], &h_hash[22], &h_hash[23], &h_hash[24], &h_hash[25], &h_hash[26], &h_hash[27], &h_hash[28], &h_hash[29], &h_hash[30], &h_hash[31]);
	h_hash[32] = '\0';
	
	char *h_message = (char *) malloc(max_str_size);
	int h_done[1] = {0};
	uint64_t starting_tid = 0;
	
	int *d_done;
	unsigned char *d_hash;
	char *d_message;
	
	hipMalloc((void**) &d_done, sizeof(int));
	hipMalloc((void**) &d_hash, digest_str_size);
	hipMalloc((void**) &d_message, max_str_size);

	hipEventRecord(start, 0);
	hipMemcpy(d_done, h_done, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_hash, h_hash, digest_size_bytes, hipMemcpyHostToDevice);
	hipEventRecord(stop, 0);
	hipEventSynchronize(start);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&h_to_d_time, start, stop);
	
	hipEventRecord(start, 0);
	while (!h_done[0]) {
		brute_force_single<<<number_blocks, number_threads>>>(d_hash, d_message, d_done, starting_tid);
		starting_tid += number_blocks * number_threads;
		hipMemcpy(h_done, d_done, sizeof(int), hipMemcpyDeviceToHost);
		
		hipError_t cudaerr = hipDeviceSynchronize();
		if (cudaerr != hipSuccess) {
			h_done[0] = 1;
			printf("kernel launch failed with error \"%s\".\n", hipGetErrorString(cudaerr));
		}
	}
	hipEventRecord(stop, 0);
    hipEventSynchronize(start);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&comp_time, start, stop);
	
	hipEventRecord(start, 0);
	hipMemcpy(h_message, d_message, max_str_size, hipMemcpyDeviceToHost);
	hipEventRecord(stop, 0);
	hipEventSynchronize(start);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&d_to_h_time, start, stop);
	
	h_to_d_time /= 1000;
	comp_time /= 1000;
	d_to_h_time /= 1000;
	total_time = h_to_d_time + comp_time + d_to_h_time;
	
	printf("Message is %s.\n", h_message);
	printf("Memory Transfer Time Host -> Device\temp%0.3g sec\n", h_to_d_time);
	printf("Computation Time\temp\temp\temp%0.3g sec\n", comp_time);
	printf("Memory Transfer Time Device -> Host\temp%0.3g sec\n", d_to_h_time);
	printf("Total Time\temp\temp\temp\temp%0.3g sec\n", total_time);
}

/**
 * Main method, initializes the global variables, calls the kernels, and prints the results.
 */
int main(int argc, char **argv)
{
    char *file_name;
	if (argc != 3)
	{
		num_messages = 10000;
		file_name = "messages.txt";
	}
	else
	{
		file_name = argv[1];
		num_messages = atoi(argv[2]);
	}
	
    gpu_init();
    hipEventCreate(&start);
    hipEventCreate(&stop);
	
	runBenchmarks(file_name);
	find_message();
	
	hipEventDestroy(start);
    hipEventDestroy(stop);

    return EXIT_SUCCESS;
}